#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "constants.h"
#include <vector>

#include <cstdio>
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void addVector(double *arrays, double *result, int size) {
    int x = blockIdx.x;

    if (x < size)
        result[x] = arrays[x] + arrays[x + size];
}

void callCudaFree()
{
    HANDLE_ERROR(hipFree(0));
}

void call_cuda_kernel(std::vector<double> const &arrays, std::vector<double> &result)
{
    double *dev_array, *dev_result;

    HANDLE_ERROR(hipMalloc((void**)&dev_array, arrays.size() * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_result, result.size() * sizeof(double)));

    HANDLE_ERROR(hipMemcpy(dev_array, arrays.data(), arrays.size() * sizeof(double), hipMemcpyHostToDevice));

    addVector<<<result.size(),1>>>(dev_array, dev_result, result.size());

    HANDLE_ERROR(hipMemcpy(result.data(), dev_result, result.size() * sizeof(double), hipMemcpyDeviceToHost));

    hipFree(dev_array);
    hipFree(dev_result);
}
