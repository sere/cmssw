#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "constants.h"
#include <vector>

#include <cstdio>
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void addVector(double *arrays, double *result) {
    int x = blockIdx.x;

    if (x < ARRAY_SIZE)
        result[x] = arrays[x] + arrays[x + ARRAY_SIZE];
}

void callCudaFree()
{
    HANDLE_ERROR(hipFree(0));
}

void call_cuda_kernel(std::vector<double> const &arrays, std::vector<double> &result)
{
    double *dev_array, *dev_result;

    HANDLE_ERROR(hipMalloc((void**)&dev_array, MATR_SIZE * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_result, ARRAY_SIZE * sizeof(double)));

    HANDLE_ERROR(hipMemcpy(dev_array, arrays.data(), MATR_SIZE * sizeof(double), hipMemcpyHostToDevice));

    addVector<<<ARRAY_SIZE,1>>>(dev_array, dev_result);

    HANDLE_ERROR(hipMemcpy(result.data(), dev_result, ARRAY_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    hipFree(dev_array);
    hipFree(dev_result);
}
