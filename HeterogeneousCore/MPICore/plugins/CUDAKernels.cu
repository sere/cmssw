#include "hip/hip_runtime.h"
#include "constants.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <cstdio>
static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void addVector(double *arrays, double *result, int size) {
    int x = blockIdx.x;

    if (x < size)
        result[x] = arrays[x] + arrays[x + size];
}

void callCudaFree() { HANDLE_ERROR(hipFree(0)); }

void allocate_buffers(double *&dev_array, double *&dev_result) {
    HANDLE_ERROR(hipMalloc(&dev_array, MAX_ARRAY_SIZE * 2 * sizeof(double)));
    HANDLE_ERROR(hipMalloc(&dev_result, MAX_ARRAY_SIZE * sizeof(double)));
}

void call_cuda_kernel(std::vector<double> const &arrays,
                      std::vector<double> &result, double *dev_array,
                      double *dev_result) {
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipMemcpy(dev_array, arrays.data(),
                            arrays.size() * sizeof(double), hipMemcpyDefault));

    addVector<<<result.size(), 1>>>(dev_array, dev_result, result.size());

    HANDLE_ERROR(hipMemcpy(result.data(), dev_result,
                            result.size() * sizeof(double),
                            hipMemcpyDeviceToHost));
}
